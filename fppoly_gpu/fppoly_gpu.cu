#include "hip/hip_runtime.h"
/*
 *
 *  This source file is part of ELINA (ETH LIbrary for Numerical Analysis).
 *  ELINA is Copyright © 2018 Department of Computer Science, ETH Zurich
 *  This software is distributed under GNU Lesser General Public License
 * Version 3.0. For more information, see the ELINA project website at:
 *  http://elina.ethz.ch
 *
 *  THE SOFTWARE IS PROVIDED "AS-IS" WITHOUT ANY WARRANTY OF ANY KIND, EITHER
 *  EXPRESS, IMPLIED OR STATUTORY, INCLUDING BUT NOT LIMITED TO ANY WARRANTY
 *  THAT THE SOFTWARE WILL CONFORM TO SPECIFICATIONS OR BE ERROR-FREE AND ANY
 *  IMPLIED WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE,
 *  TITLE, OR NON-INFRINGEMENT.  IN NO EVENT SHALL ETH ZURICH BE LIABLE FOR ANY
 *  DAMAGES, INCLUDING BUT NOT LIMITED TO DIRECT, INDIRECT,
 *  SPECIAL OR CONSEQUENTIAL DAMAGES, ARISING OUT OF, RESULTING FROM, OR IN
 *  ANY WAY CONNECTED WITH THIS SOFTWARE (WHETHER OR NOT BASED UPON WARRANTY,
 *  CONTRACT, TORT OR OTHERWISE).
 *
 */

#include "fppoly_gpu.h"

#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>

const size_t num_threads = 128;

bool results[90];
bool results_calculated;
size_t output_counter;

#ifdef single
__constant__ const float_type min_denormal = 1.40129846e-45;
__constant__ const float_type ulp = 1.1920929e-07;
#else
__constant__ const float_type min_denormal = 4.940656458412465441766e-324;
__constant__ const float_type ulp = 2.220446049250313080848e-16;
#endif

#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

__device__ void
elina_double_interval_mul(float_type *const a_inf, float_type *const a_sup,
                          const float_type b_inf, const float_type b_sup,
                          const float_type c_inf, const float_type c_sup) {
  if (c_inf <= 0) {
    /* interval c is positive */
    if (b_inf <= 0) {
      /*interval b is positive*/
      if ((b_inf == 0) || (c_inf == 0)) {
        *a_inf = 0.0;
      } else {
        *a_inf = b_inf * -c_inf;
      }

      if ((b_sup == 0) || (c_sup == 0)) {
        *a_sup = 0.0;
      } else {
        *a_sup = b_sup * c_sup;
      }
    } else if (b_sup <= 0) {
      /* interval b is negative */
      if ((c_sup == 0) || (b_inf == 0)) {
        *a_inf = 0.0;
      } else {
        *a_inf = c_sup * b_inf;
      }

      if ((c_inf == 0) || (b_sup == 0)) {
        *a_sup = 0.0;
      } else {
        *a_sup = -c_inf * b_sup;
      }
    } else {
      /* there is 0 in between for b */
      if ((c_sup == 0) || (b_inf == 0)) {
        *a_inf = 0.0;
      } else {
        *a_inf = b_inf * c_sup;
      }

      if ((c_sup == 0) || (b_sup == 0)) {
        *a_sup = 0.0;
      } else {
        *a_sup = b_sup * c_sup;
      }
    }
  } else if (c_sup <= 0) {
    /* interval c is negative */
    if (b_inf <= 0) {
      /*interval b is positive*/
      if ((b_sup == 0) || (c_inf == 0)) {
        *a_inf = 0.0;
      } else {
        *a_inf = b_sup * c_inf;
      }

      if ((b_inf == 0) || (c_sup == 0)) {
        *a_sup = 0.0;
      } else {
        *a_sup = -b_inf * c_sup;
      }
    } else if (b_sup <= 0) {
      /* interval b is negative */
      if ((b_sup == 0) || (c_sup == 0)) {
        *a_inf = 0.0;
      } else {
        *a_inf = b_sup * -c_sup;
      }

      if ((b_inf == 0) || (c_inf == 0)) {
        *a_sup = 0.0;
      } else {
        *a_sup = b_inf * c_inf;
      }
    } else {
      /* there is 0 in between for b */
      if ((c_inf == 0) || (b_sup == 0)) {
        *a_inf = 0.0;
      } else {
        *a_inf = b_sup * c_inf;
      }

      if ((c_inf == 0) || (b_inf == 0)) {
        *a_sup = 0.0;
      } else {
        *a_sup = b_inf * c_inf;
      }
    }
  } else if (b_inf <= 0) {
    /* interval b is positive */
    if (c_inf <= 0) {
      /*interval c is positive */
      if ((b_inf == 0) || (c_inf == 0)) {
        *a_inf = 0.0;
      } else {
        *a_inf = -b_inf * c_inf;
      }

      if ((b_sup == 0) || (c_sup == 0)) {
        *a_sup = 0.0;
      } else {
        *a_sup = b_sup * c_sup;
      }
    } else if (c_sup <= 0) {
      /* interval c is negative */
      if ((b_sup == 0) || (c_inf == 0)) {
        *a_inf = 0.0;
      } else {
        *a_inf = b_sup * c_inf;
      }

      if ((b_inf == 0) || (c_sup == 0)) {
        *a_sup = 0.0;
      } else {
        *a_sup = -b_inf * c_sup;
      }
    } else {
      /* there is 0 in between for c */
      if ((b_sup == 0) || (c_inf == 0)) {
        *a_inf = 0.0;
      } else {
        *a_inf = b_sup * c_inf;
      }

      if ((b_sup == 0) || (c_sup == 0)) {
        *a_sup = 0.0;
      } else {
        *a_sup = b_sup * c_sup;
      }
    }
  } else if (b_sup <= 0) {
    /* interval b is negative */
    if (c_inf <= 0) {
      /* interval c is positive */
      if ((b_inf == 0) || (c_sup == 0)) {
        *a_inf = 0.0;
      } else {
        *a_inf = b_inf * c_sup;
      }

      if ((b_sup == 0) || (c_inf == 0)) {
        *a_sup = 0.0;
      } else {
        *a_sup = b_sup * -c_inf;
      }
    } else if (c_sup <= 0) {
      /* interval c is negative */
      if ((b_sup == 0) || (c_sup == 0)) {
        *a_inf = 0.0;
      } else {
        *a_inf = -b_sup * c_sup;
      }

      if ((b_inf == 0) || (c_inf == 0)) {
        *a_sup = 0.0;
      } else {
        *a_sup = b_inf * c_inf;
      }
    } else {
      /* there is 0 in between for c */
      if ((b_inf == 0) || (c_sup == 0)) {
        *a_inf = 0.0;
      } else {
        *a_inf = b_inf * c_sup;
      }

      if ((b_inf == 0) || (c_inf == 0)) {
        *a_sup = 0.0;
      } else {
        *a_sup = b_inf * c_inf;
      }
    }
  } else {
    /* there is 0 in between for both b and c */
    float_type tmp_inf1 = b_sup * c_inf;
    float_type tmp_sup1 = b_inf * c_inf;
    float_type tmp_inf2 = b_inf * c_sup;
    float_type tmp_sup2 = b_sup * c_sup;
    *a_inf = fmax(tmp_inf1, tmp_inf2);
    *a_sup = fmax(tmp_sup1, tmp_sup2);
  }
}

__device__ void
elina_double_interval_div(float_type *const a_inf, float_type *const a_sup,
                          const float_type b_inf, const float_type b_sup,
                          const float_type c_inf, const float_type c_sup) {
  if (c_inf < 0) {
    /* c is positive */
    if (b_inf <= 0) {
      /* b is positive */
      *a_inf = b_inf / c_sup;
      *a_sup = b_sup / -c_inf;
    } else if (b_sup <= 0) {
      /* b is negative */
      *a_inf = -b_inf / c_inf;
      *a_sup = b_sup / c_sup;
    } else {
      /* 0 is in the middle of b: one divides b by c->inf */
      *a_inf = b_inf / -c_inf;
      *a_sup = b_sup / -c_inf;
    }
  } else if (c_sup < 0) {
    /* c is negative */
    if (b_inf <= 0) {
      /* b is positive */
      *a_sup = b_inf / c_inf;
      *a_inf = -b_sup / c_sup;
    } else if (b_sup <= 0) {
      /* b is negative */
      *a_inf = b_sup / c_inf;
      *a_sup = -b_inf / c_sup;
    } else {
      /* 0 is in the middle of b: one cross-divide b by c->sup */
      *a_inf = b_sup / c_sup;
      *a_sup = b_inf / c_sup;
    }
  } else if ((b_inf == 0) && (b_sup == 0)) {
    /* b is [0,0] */
    *a_inf = b_inf;
    *a_sup = b_sup;
  } else {
    *a_inf = INFINITY;
    *a_sup = INFINITY;
  }
}

fppoly_t *fppoly_of_abstract0(elina_abstract0_t *a) {
  return (fppoly_t *)a->value;
}

elina_abstract0_t *abstract0_of_fppoly(elina_manager_t *man, fppoly_t *fp) {
  elina_abstract0_t *r = (elina_abstract0_t *)malloc(sizeof(elina_abstract0_t));
  assert(r);
  r->value = fp;
  r->man = elina_manager_copy(man);

  return r;
}

static inline void fppoly_internal_free(fppoly_internal_t *pr) {
  if (pr) {
    pr->funid = ELINA_FUNID_UNKNOWN;
    free(pr);
    pr = nullptr;
  }
}

static inline fppoly_internal_t *fppoly_internal_alloc() {
  fppoly_internal_t *pr =
      (fppoly_internal_t *)malloc(sizeof(fppoly_internal_t));
  pr->funid = ELINA_FUNID_UNKNOWN;
  pr->man = nullptr;
  pr->funopt = nullptr;
  pr->min_denormal = ldexpl(1.0, -1074);
  pr->ulp = ldexpl(1.0, -52);

  return pr;
}

/* back pointer to our internal structure from the manager */
fppoly_internal_t *fppoly_init_from_manager(elina_manager_t *man,
                                            elina_funid_t funid) {
  fppoly_internal_t *pr = (fppoly_internal_t *)man->internal;
  pr->funid = funid;

  if (!(pr->man)) {
    pr->man = man;
  }

  return pr;
}

elina_manager_t *fppoly_manager_alloc() {
  std::cout << "This is the GPU version of fppoly!" << std::endl;
  results_calculated = false;
  output_counter = 1;

  void **funptr;
  fppoly_internal_t *pr = fppoly_internal_alloc();

  elina_manager_t *man = elina_manager_alloc(
      "fppoly",                              /* Library name */
      "1.0",                                 /* version */
      pr,                                    /* internal structure */
      (void (*)(void *))fppoly_internal_free /* free function for internal */
  );

  funptr = man->funptr;
  funptr[ELINA_FUNID_FREE] = (void *)&fppoly_free;
  /* 3.Printing */
  funptr[ELINA_FUNID_FPRINT] = (void *)&fppoly_fprint;

  return man;
}

/*
__device__
void expr_print(const expr_t* const expr)
{
    if((expr->inf_coeff == nullptr) || (expr->sup_coeff == nullptr))
    {
        printf("+ [%g, %g]\n", -expr->inf_cst, expr->sup_cst);

        return;
    }

    for(size_t i = 0; i < size; i++)
    {
        if(i == 0)
        {
            printf("[%g, %g]x0 ", -expr->inf_coeff[0], expr->sup_coeff[0]);
        }
        else
        {
            printf("+ [%g, %g]x%zu ", -expr->inf_coeff[i], expr->sup_coeff[i],
i);
        }
    }

    printf("+ [%g, %g]\n", -expr->inf_cst, expr->sup_cst);
}
*/

layer_t *create_layer(const size_t num_out_neurons, const size_t num_in_neurons,
                      const layertype_t type,
                      const activation_type_t activation) {
  layer_t *layer = (layer_t *)malloc(sizeof(layer_t));

  layer->num_out_neurons = num_out_neurons;
  layer->num_in_neurons = num_in_neurons;

  layer->type = type;
  layer->activation = activation;

  hipMalloc((void **)&layer->lb_array, num_out_neurons * sizeof(float_type));
  hipMalloc((void **)&layer->ub_array, num_out_neurons * sizeof(float_type));

  hipMalloc((void **)&layer->inf_coeff,
             num_out_neurons * num_in_neurons * sizeof(float_type));
  hipMalloc((void **)&layer->sup_coeff,
             num_out_neurons * num_in_neurons * sizeof(float_type));

  hipMalloc((void **)&layer->inf_cst, num_out_neurons * sizeof(float_type));
  hipMalloc((void **)&layer->sup_cst, num_out_neurons * sizeof(float_type));

  return layer;
}

void fppoly_from_network_input_box(fppoly_t *const res, const size_t intdim,
                                   const size_t realdim,
                                   const double *inf_array,
                                   const double *sup_array) {
  res->layers = nullptr;
  res->numlayers = 0;

  size_t num_pixels = intdim + realdim;

  float_type *tmp_input_inf =
      (float_type *)malloc(num_pixels * sizeof(float_type));
  float_type *tmp_input_sup =
      (float_type *)malloc(num_pixels * sizeof(float_type));

  for (size_t i = 0; i < num_pixels; i++) {
    tmp_input_inf[i] = -inf_array[i];
    tmp_input_sup[i] = sup_array[i];
  }

  hipMalloc((void **)&(res->input_inf), num_pixels * sizeof(float_type));
  hipMalloc((void **)&(res->input_sup), num_pixels * sizeof(float_type));

  hipMemcpy(res->input_inf, tmp_input_inf, num_pixels * sizeof(float_type),
             hipMemcpyHostToDevice);
  hipMemcpy(res->input_sup, tmp_input_sup, num_pixels * sizeof(float_type),
             hipMemcpyHostToDevice);

  free(tmp_input_inf);
  free(tmp_input_sup);

  res->num_pixels = num_pixels;
}

elina_abstract0_t *fppoly_from_network_input(elina_manager_t *man,
                                             const size_t intdim,
                                             const size_t realdim,
                                             const double *inf_array,
                                             const double *sup_array) {
  fppoly_t *res = (fppoly_t *)malloc(sizeof(fppoly_t));
  fppoly_from_network_input_box(res, intdim, realdim, inf_array, sup_array);

  return abstract0_of_fppoly(man, res);
}

void fppoly_add_new_layer(fppoly_t *const fp, const size_t num_out_neurons,
                          const size_t num_in_neurons, const layertype_t type,
                          const activation_type_t activation) {
  const size_t numlayers = fp->numlayers;
  fp->layers[numlayers] =
      create_layer(num_out_neurons, num_in_neurons, type, activation);
  fp->numlayers++;
}

__device__ void elina_double_interval_add_expr_coeff(
    float_type *const res_inf, float_type *const res_sup, const float_type inf,
    const float_type sup, const float_type inf_expr,
    const float_type sup_expr) {
  *res_inf = inf + inf_expr;
  *res_sup = sup + sup_expr;
  const float_type maxA = fmax(fabs(inf_expr), fabs(sup_expr));
  float_type tmp1, tmp2;
  elina_double_interval_mul(&tmp1, &tmp2, inf, sup, maxA * ulp, maxA * ulp);
  *res_inf += tmp1;
  *res_sup += tmp2;
}

__device__ void elina_double_interval_add_cst_coeff(float_type *const res_inf,
                                                    float_type *const res_sup,
                                                    const float_type inf,
                                                    const float_type sup,
                                                    const float_type inf_expr,
                                                    const float_type sup_expr) {
  elina_double_interval_add_expr_coeff(res_inf, res_sup, inf, sup, inf_expr,
                                       sup_expr);
  *res_inf += min_denormal;
  *res_sup += min_denormal;
}

__device__ void elina_double_interval_mul_expr_coeff(
    float_type *const res_inf, float_type *const res_sup, const float_type inf,
    const float_type sup, const float_type inf_expr,
    const float_type sup_expr) {
  elina_double_interval_mul(res_inf, res_sup, inf, sup, inf_expr, sup_expr);
  const float_type maxA = fmax(fabs(inf_expr), fabs(sup_expr));
  float_type tmp1, tmp2;
  elina_double_interval_mul(&tmp1, &tmp2, inf, sup, maxA * ulp, maxA * ulp);
  *res_inf += tmp1;
  *res_sup += tmp2;
}

__device__ void elina_double_interval_mul_cst_coeff(float_type *const res_inf,
                                                    float_type *const res_sup,
                                                    const float_type inf,
                                                    const float_type sup,
                                                    const float_type inf_expr,
                                                    const float_type sup_expr) {
  elina_double_interval_mul_expr_coeff(res_inf, res_sup, inf, sup, inf_expr,
                                       sup_expr);
  *res_inf += min_denormal;
  *res_sup += min_denormal;
}

__global__ void compute_lb_from_expr(float_type *__restrict__ lb_array,
                                     const float_type *__restrict__ inf_coeff,
                                     const float_type *__restrict__ sup_coeff,
                                     const float_type *__restrict__ inf_cst,
                                     const float_type *__restrict__ input_inf,
                                     const float_type *__restrict__ input_sup,
                                     const size_t num_exprs,
                                     const size_t expr_size) {
  const size_t n = blockIdx.x;

  float_type res_inf = inf_cst[n];

  float_type tmp1, tmp2;

  for (size_t i = 0; i < expr_size; i++) {
    elina_double_interval_mul(&tmp1, &tmp2, inf_coeff[n * expr_size + i],
                              sup_coeff[n * expr_size + i], input_inf[i],
                              input_sup[i]);
    res_inf = res_inf + tmp1;
  }

  lb_array[n] = res_inf;
}

__global__ void compute_ub_from_expr(float_type *__restrict__ ub_array,
                                     const float_type *__restrict__ inf_coeff,
                                     const float_type *__restrict__ sup_coeff,
                                     const float_type *__restrict__ sup_cst,
                                     const float_type *__restrict__ input_inf,
                                     const float_type *__restrict__ input_sup,
                                     const size_t num_exprs,
                                     const size_t expr_size) {
  const size_t n = blockIdx.x;

  float_type res_sup = sup_cst[n];

  float_type tmp1, tmp2;

  for (size_t i = 0; i < expr_size; i++) {
    elina_double_interval_mul(&tmp1, &tmp2, inf_coeff[n * expr_size + i],
                              sup_coeff[n * expr_size + i], input_inf[i],
                              input_sup[i]);
    res_sup = res_sup + tmp2;
  }

  ub_array[n] = res_sup;
}

__global__ void device_layer_create_dense_expr(
    float_type *__restrict__ inf_coeff, float_type *__restrict__ sup_coeff,
    float_type *__restrict__ inf_cst, float_type *__restrict__ sup_cst,
    const double *__restrict__ weights, const double *__restrict__ bias,
    const size_t num_out_neurons, const size_t num_in_neurons) {
  const size_t i = blockIdx.x;

  const double *weight_i = weights + i * num_in_neurons;
  const double bias_i = bias[i];

  inf_cst[i] = -bias_i;
  sup_cst[i] = bias_i;

  for (size_t j = 0; j < num_in_neurons; j++) {
    inf_coeff[i * num_in_neurons + j] = -weight_i[j];
    sup_coeff[i * num_in_neurons + j] = weight_i[j];
  }
}

void layer_create_dense_exprs(float_type *inf_coeff, float_type *sup_coeff,
                              float_type *inf_cst, float_type *sup_cst,
                              const double **weights, const double *bias,
                              const size_t num_out_neurons,
                              const size_t num_in_neurons) {
  double *tmp_weights;
  hipMalloc((void **)&tmp_weights,
             num_out_neurons * num_in_neurons * sizeof(double));

  double *tmp_bias;
  hipMalloc((void **)&tmp_bias, num_out_neurons * sizeof(double));

  for (size_t i = 0; i < num_out_neurons; i++) {
    hipMemcpy(tmp_weights + i * num_in_neurons, weights[i],
               num_in_neurons * sizeof(double), hipMemcpyHostToDevice);
  }

  hipMemcpy(tmp_bias, bias, num_out_neurons * sizeof(double),
             hipMemcpyHostToDevice);

  device_layer_create_dense_expr<<<num_out_neurons, 1>>>(
      inf_coeff, sup_coeff, inf_cst, sup_cst, tmp_weights, tmp_bias,
      num_out_neurons, num_in_neurons);

  hipFree(tmp_weights);
  hipFree(tmp_bias);
}

__global__ void copy_expr_array(float_type *__restrict__ target_inf_coeff,
                                float_type *__restrict__ target_sup_coeff,
                                float_type *__restrict__ target_inf_cst,
                                float_type *__restrict__ target_sup_cst,
                                const float_type *__restrict__ source_inf_coeff,
                                const float_type *__restrict__ source_sup_coeff,
                                const float_type *__restrict__ source_inf_cst,
                                const float_type *__restrict__ source_sup_cst,
                                const size_t num_exprs,
                                const size_t expr_size) {
  const size_t i = blockIdx.x;

  for (size_t j = 0; j < expr_size; j++) {
    target_inf_coeff[i * expr_size + j] = source_inf_coeff[i * expr_size + j];
    target_sup_coeff[i * expr_size + j] = source_sup_coeff[i * expr_size + j];
  }

  target_inf_cst[i] = source_inf_cst[i];
  target_sup_cst[i] = source_sup_cst[i];
}

void layer_compute_bounds_from_exprs(
    float_type *inf_coeff, float_type *sup_coeff, float_type *inf_cst,
    float_type *sup_cst, float_type *lb_array, float_type *ub_array,
    float_type *input_inf, float_type *input_sup, const size_t num_out_neurons,
    const size_t num_in_neurons) {
  compute_lb_from_expr<<<num_out_neurons, 1>>>(lb_array, inf_coeff, sup_coeff,
                                               inf_cst, input_inf, input_sup,
                                               num_out_neurons, num_in_neurons);
  compute_ub_from_expr<<<num_out_neurons, 1>>>(ub_array, inf_coeff, sup_coeff,
                                               sup_cst, input_inf, input_sup,
                                               num_out_neurons, num_in_neurons);
}

void ffn_handle_first_layer(elina_manager_t *man, elina_abstract0_t *abs,
                            const double **weights, const double *bias,
                            const size_t size, const size_t num_pixels,
                            const activation_type_t activation) {
  fppoly_t *res = fppoly_of_abstract0(abs);
  fppoly_internal_t *pr =
      fppoly_init_from_manager(man, ELINA_FUNID_ASSIGN_LINEXPR_ARRAY);

  res->layers = (layer_t **)malloc(20 * sizeof(layer_t *));
  fppoly_add_new_layer(res, size, num_pixels, FFN, activation);

  float_type *inf_coeff = res->layers[0]->inf_coeff;
  float_type *sup_coeff = res->layers[0]->sup_coeff;

  float_type *inf_cst = res->layers[0]->inf_cst;
  float_type *sup_cst = res->layers[0]->sup_cst;

  layer_create_dense_exprs(inf_coeff, sup_coeff, inf_cst, sup_cst, weights,
                           bias, size, num_pixels);
  layer_compute_bounds_from_exprs(
      inf_coeff, sup_coeff, inf_cst, sup_cst, res->layers[0]->lb_array,
      res->layers[0]->ub_array, res->input_inf, res->input_sup,
      res->layers[0]->num_out_neurons, res->layers[0]->num_in_neurons);
}

void ffn_handle_first_relu_layer(elina_manager_t *man, elina_abstract0_t *abs,
                                 const double **weights, const double *bias,
                                 const size_t size, const size_t num_pixels) {
  ffn_handle_first_layer(man, abs, weights, bias, size, num_pixels, RELU);
}

void ffn_handle_first_sigmoid_layer(elina_manager_t *man,
                                    elina_abstract0_t *abs,
                                    const double **weights, const double *bias,
                                    const size_t size,
                                    const size_t num_pixels) {
  // ffn_handle_first_layer(man, abs, weights, bias, size, num_pixels, SIGMOID);
}

void ffn_handle_first_tanh_layer(elina_manager_t *man, elina_abstract0_t *abs,
                                 const double **weights, const double *bias,
                                 const size_t size, const size_t num_pixels) {
  // ffn_handle_first_layer(man, abs, weights, bias, size, num_pixels, TANH);
}

__global__ void lexpr_replace_relu_bounds(
    float_type *__restrict__ inf_coeff, float_type *__restrict__ sup_coeff,
    float_type *__restrict__ inf_cst, float_type *__restrict__ sup_cst,
    const float_type *__restrict__ lb_array,
    const float_type *__restrict__ ub_array,
    const size_t num_out_neurons_last_layer,
    const size_t num_out_neurons_current_layer) {
  const size_t n = blockIdx.x;
  const size_t i = blockIdx.y * blockDim.x + threadIdx.x;

  if (i < num_out_neurons_current_layer) {
    const size_t a = n * num_out_neurons_current_layer + i;

    const float_type lb = lb_array[i];
    const float_type ub = ub_array[i];
    const float_type width = ub + lb;
    const float_type lambda_inf = -ub / width;
    const float_type lambda_sup = ub / width;

    const float_type old_inf_coeff = inf_coeff[a];
    const float_type old_sup_coeff = sup_coeff[a];

    if ((old_sup_coeff == 0) && (old_inf_coeff == 0)) {
      inf_coeff[a] = 0.0;
      sup_coeff[a] = 0.0;

      return;
    } else if (ub <= 0) {
      inf_coeff[a] = 0.0;
      sup_coeff[a] = 0.0;

      return;
    } else if (lb < 0) {
      inf_coeff[a] = old_inf_coeff;
      sup_coeff[a] = old_sup_coeff;
    } else if (old_sup_coeff < 0) {
      const float_type mu_inf = lambda_inf * lb;
      const float_type mu_sup = lambda_sup * lb;
      elina_double_interval_mul_expr_coeff(&inf_coeff[a], &sup_coeff[a],
                                           lambda_inf, lambda_sup,
                                           old_inf_coeff, old_sup_coeff);
      float_type tmp1, tmp2;
      elina_double_interval_mul_cst_coeff(&tmp1, &tmp2, mu_inf, mu_sup,
                                          old_inf_coeff, old_sup_coeff);

      atomicAdd(&inf_cst[n], tmp1 + min_denormal);
      atomicAdd(&sup_cst[n], tmp2 + min_denormal);
    } else if (old_inf_coeff < 0) {
      const float_type area1 = lb * ub;
      const float_type area2 = 0.5 * ub * width;
      const float_type area3 = 0.5 * lb * width;

      if ((area1 < area2) && (area1 < area3)) {
        elina_double_interval_mul_expr_coeff(&inf_coeff[a], &sup_coeff[a],
                                             lambda_inf, lambda_sup,
                                             old_inf_coeff, old_sup_coeff);
      } else if ((area2 < area1) && (area2 < area3)) {
        inf_coeff[a] = 0.0;
        sup_coeff[a] = 0.0;
      } else {
        inf_coeff[a] = old_inf_coeff;
        sup_coeff[a] = old_sup_coeff;
      }
    } else {
      inf_coeff[a] = 0.0;
      sup_coeff[a] = 0.0;
      float_type tmp1, tmp2;
      elina_double_interval_mul(&tmp1, &tmp2, old_inf_coeff, old_sup_coeff, 0,
                                ub);

      atomicAdd(&inf_cst[n], tmp1);
      atomicAdd(&sup_cst[n], -tmp1);
    }
  }
}

__global__ void uexpr_replace_relu_bounds(
    float_type *__restrict__ inf_coeff, float_type *__restrict__ sup_coeff,
    float_type *__restrict__ inf_cst, float_type *__restrict__ sup_cst,
    const float_type *__restrict__ lb_array,
    const float_type *__restrict__ ub_array,
    const size_t num_out_neurons_last_layer,
    const size_t num_out_neurons_current_layer) {
  const size_t n = blockIdx.x;
  const size_t i = blockIdx.y * blockDim.x + threadIdx.x;

  if (i < num_out_neurons_current_layer) {
    const size_t a = n * num_out_neurons_current_layer + i;

    const float_type lb = lb_array[i];
    const float_type ub = ub_array[i];
    const float_type width = ub + lb;
    const float_type lambda_inf = -ub / width;
    const float_type lambda_sup = ub / width;

    const float_type old_inf_coeff = inf_coeff[a];
    const float_type old_sup_coeff = sup_coeff[a];

    if ((old_sup_coeff == 0) && (old_inf_coeff == 0)) {
      inf_coeff[a] = 0.0;
      sup_coeff[a] = 0.0;

      return;
    } else if (ub <= 0) {
      inf_coeff[a] = 0.0;
      sup_coeff[a] = 0.0;

      return;
    } else if (lb < 0) {
      inf_coeff[a] = old_inf_coeff;
      sup_coeff[a] = old_sup_coeff;
    } else if (old_inf_coeff < 0) {
      const float_type mu_inf = lambda_inf * lb;
      const float_type mu_sup = lambda_sup * lb;
      elina_double_interval_mul_expr_coeff(&inf_coeff[a], &sup_coeff[a],
                                           lambda_inf, lambda_sup,
                                           old_inf_coeff, old_sup_coeff);
      float_type tmp1, tmp2;
      elina_double_interval_mul_cst_coeff(&tmp1, &tmp2, mu_inf, mu_sup,
                                          old_inf_coeff, old_sup_coeff);

      atomicAdd(&inf_cst[n], tmp1 + min_denormal);
      atomicAdd(&sup_cst[n], tmp2 + min_denormal);
    } else if (old_sup_coeff < 0) {
      const float_type area1 = lb * ub;
      const float_type area2 = 0.5 * ub * width;
      const float_type area3 = 0.5 * lb * width;

      if ((area1 < area2) && (area1 < area3)) {
        elina_double_interval_mul_expr_coeff(&inf_coeff[a], &sup_coeff[a],
                                             lambda_inf, lambda_sup,
                                             old_inf_coeff, old_sup_coeff);
      } else if ((area2 < area1) && (area2 < area3)) {
        inf_coeff[a] = 0.0;
        sup_coeff[a] = 0.0;
      } else {
        inf_coeff[a] = old_inf_coeff;
        sup_coeff[a] = old_sup_coeff;
      }
    } else {
      inf_coeff[a] = 0.0;
      sup_coeff[a] = 0.0;
      float_type tmp1, tmp2;
      elina_double_interval_mul(&tmp1, &tmp2, old_inf_coeff, old_sup_coeff, 0,
                                ub);

      atomicAdd(&inf_cst[n], -tmp2);
      atomicAdd(&sup_cst[n], tmp2);
    }
  }
}

// TODO: Try to load values from aux-array only once and use them multiple
// times!
__global__ void
coeffs_from_previous_layer(const float_type *__restrict__ expr_inf_coeff,
                           const float_type *__restrict__ expr_sup_coeff,
                           float_type *__restrict__ res_inf_coeff,
                           float_type *__restrict__ res_sup_coeff,
                           const float_type *__restrict__ aux_inf_coeff,
                           const float_type *__restrict__ aux_sup_coeff,
                           const size_t num_out_neurons_last_layer,
                           const size_t num_out_neurons_current_layer,
                           const size_t num_in_neurons_current_layer) {
  const size_t n = blockIdx.x;
  const size_t j = blockIdx.y * blockDim.x + threadIdx.x;

  if (j < num_in_neurons_current_layer) {
    size_t i = 0;

    size_t a = n * num_out_neurons_current_layer + i;
    const size_t b = n * num_in_neurons_current_layer + j;
    size_t c = i * num_in_neurons_current_layer + j;

    float_type inf_coeff;
    float_type sup_coeff;

    elina_double_interval_mul_expr_coeff(&inf_coeff, &sup_coeff,
                                         expr_inf_coeff[a], expr_sup_coeff[a],
                                         aux_inf_coeff[c], aux_sup_coeff[c]);

    float_type tmp1, tmp2;
    float_type maxRes, maxMul;

    for (i = 1; i < num_out_neurons_current_layer; i++) {
      a++;
      c += num_in_neurons_current_layer;

      const float_type prev_inf_coeff = expr_inf_coeff[a];
      const float_type prev_sup_coeff = expr_sup_coeff[a];

      if ((prev_inf_coeff != 0) || (prev_sup_coeff != 0)) {
        elina_double_interval_mul_expr_coeff(&tmp1, &tmp2, prev_inf_coeff,
                                             prev_sup_coeff, aux_inf_coeff[c],
                                             aux_sup_coeff[c]);

        maxRes = fmax(fabs(inf_coeff), fabs(sup_coeff));
        maxMul = fmax(fabs(tmp1), fabs(tmp2));

        inf_coeff = inf_coeff + tmp1 + (maxRes + maxMul) * ulp;
        sup_coeff = sup_coeff + tmp2 + (maxRes + maxMul) * ulp;
      }
    }

    res_inf_coeff[b] = inf_coeff;
    res_sup_coeff[b] = sup_coeff;
  }
}

__global__ void
csts_from_previous_layer(const float_type *__restrict__ expr_inf_coeff,
                         const float_type *__restrict__ expr_sup_coeff,
                         const float_type *__restrict__ expr_inf_cst,
                         const float_type *__restrict__ expr_sup_cst,
                         float_type *__restrict__ res_inf_cst,
                         float_type *__restrict__ res_sup_cst,
                         const float_type *__restrict__ aux_inf_cst,
                         const float_type *__restrict__ aux_sup_cst,
                         const size_t num_out_neurons_last_layer,
                         const size_t num_out_neurons_current_layer) {
  const size_t n = blockIdx.x;

  size_t i = 0;

  size_t a = n * num_out_neurons_current_layer + i;

  float_type inf_cst;
  float_type sup_cst;

  elina_double_interval_mul_cst_coeff(&inf_cst, &sup_cst, expr_inf_coeff[a],
                                      expr_sup_coeff[a], aux_inf_cst[i],
                                      aux_sup_cst[i]);

  float_type tmp1, tmp2;
  float_type maxRes, maxMul;

  for (i = 1; i < num_out_neurons_current_layer; i++) {
    a++;

    const float_type prev_inf_coeff = expr_inf_coeff[a];
    const float_type prev_sup_coeff = expr_sup_coeff[a];

    if ((prev_inf_coeff != 0) || (prev_sup_coeff != 0)) {
      elina_double_interval_mul_cst_coeff(&tmp1, &tmp2, prev_inf_coeff,
                                          prev_sup_coeff, aux_inf_cst[i],
                                          aux_sup_cst[i]);

      maxRes = fmax(fabs(inf_cst), fabs(sup_cst));
      maxMul = fmax(fabs(tmp1), fabs(tmp2));

      inf_cst += tmp1 + (maxRes + maxMul) * ulp + min_denormal;
      sup_cst += tmp2 + (maxRes + maxMul) * ulp + min_denormal;
    }
  }

  res_inf_cst[n] = inf_cst + expr_inf_cst[n];
  res_sup_cst[n] = sup_cst + expr_sup_cst[n];
}

void update_state_using_previous_layers(elina_manager_t *man, fppoly_t *fp,
                                        const size_t layerno) {
  auto start = std::chrono::system_clock::now();

  fppoly_internal_t *pr =
      fppoly_init_from_manager(man, ELINA_FUNID_ASSIGN_LINEXPR_ARRAY);

  const size_t num_out_neurons_last_layer =
      fp->layers[layerno]->num_out_neurons;
  const size_t num_in_neurons_last_layer = fp->layers[layerno]->num_in_neurons;

  const size_t num_in_neurons_first_layer = fp->layers[0]->num_in_neurons;

  std::cout << "num_out_neurons_last " << num_out_neurons_last_layer
            << std::endl;

  float_type *inf_coeff = fp->layers[layerno]->inf_coeff;
  float_type *sup_coeff = fp->layers[layerno]->sup_coeff;

  float_type *inf_cst = fp->layers[layerno]->inf_cst;
  float_type *sup_cst = fp->layers[layerno]->sup_cst;

  float_type *lb_array = fp->layers[layerno]->lb_array;
  float_type *ub_array = fp->layers[layerno]->ub_array;

  float_type *linf_coeff;
  float_type *lsup_coeff;
  float_type *linf_cst;
  float_type *lsup_cst;

  hipMalloc((void **)&linf_coeff, num_out_neurons_last_layer *
                                       num_in_neurons_last_layer *
                                       sizeof(float_type));
  hipMalloc((void **)&lsup_coeff, num_out_neurons_last_layer *
                                       num_in_neurons_last_layer *
                                       sizeof(float_type));
  hipMalloc((void **)&linf_cst,
             num_out_neurons_last_layer * sizeof(float_type));
  hipMalloc((void **)&lsup_cst,
             num_out_neurons_last_layer * sizeof(float_type));

  float_type *uinf_coeff;
  float_type *usup_coeff;
  float_type *uinf_cst;
  float_type *usup_cst;

  hipMalloc((void **)&uinf_coeff, num_out_neurons_last_layer *
                                       num_in_neurons_last_layer *
                                       sizeof(float_type));
  hipMalloc((void **)&usup_coeff, num_out_neurons_last_layer *
                                       num_in_neurons_last_layer *
                                       sizeof(float_type));
  hipMalloc((void **)&uinf_cst,
             num_out_neurons_last_layer * sizeof(float_type));
  hipMalloc((void **)&usup_cst,
             num_out_neurons_last_layer * sizeof(float_type));

  copy_expr_array<<<num_out_neurons_last_layer, 1>>>(
      linf_coeff, lsup_coeff, linf_cst, lsup_cst, inf_coeff, sup_coeff, inf_cst,
      sup_cst, num_out_neurons_last_layer, num_in_neurons_last_layer);
  copy_expr_array<<<num_out_neurons_last_layer, 1>>>(
      uinf_coeff, usup_coeff, uinf_cst, usup_cst, inf_coeff, sup_coeff, inf_cst,
      sup_cst, num_out_neurons_last_layer, num_in_neurons_last_layer);

  float_type *linf_coeff_tmp;
  float_type *lsup_coeff_tmp;
  float_type *linf_cst_tmp;
  float_type *lsup_cst_tmp;

  float_type *uinf_coeff_tmp;
  float_type *usup_coeff_tmp;
  float_type *uinf_cst_tmp;
  float_type *usup_cst_tmp;

  hipMalloc((void **)&linf_cst_tmp,
             num_out_neurons_last_layer * sizeof(float_type));
  hipMalloc((void **)&lsup_cst_tmp,
             num_out_neurons_last_layer * sizeof(float_type));
  hipMalloc((void **)&uinf_cst_tmp,
             num_out_neurons_last_layer * sizeof(float_type));
  hipMalloc((void **)&usup_cst_tmp,
             num_out_neurons_last_layer * sizeof(float_type));

  for (int k = layerno - 1; k >= 0; k--) {
    const size_t num_out_neurons_current_layer = fp->layers[k]->num_out_neurons;
    const size_t num_in_neurons_current_layer = fp->layers[k]->num_in_neurons;
    std::cout << "num_out_neurons_current " << num_out_neurons_current_layer
              << " num_in_neurons_current " << num_in_neurons_current_layer
              << std::endl;

    const dim3 num_blocks_relu(num_out_neurons_last_layer,
                               num_out_neurons_current_layer / num_threads + 1,
                               1);
    const dim3 num_blocks_linear(num_out_neurons_last_layer,
                                 num_in_neurons_current_layer / num_threads + 1,
                                 1);

    std::cout << "num_threads" << num_threads << " num_blocks_relu "
              << num_blocks_relu.y << " num_blocks_linear "
              << num_blocks_linear.y << std::endl;

    float_type *aux_inf_coeff = fp->layers[k]->inf_coeff;
    float_type *aux_sup_coeff = fp->layers[k]->sup_coeff;

    float_type *aux_inf_cst = fp->layers[k]->inf_cst;
    float_type *aux_sup_cst = fp->layers[k]->sup_cst;

    float_type *aux_lb_array = fp->layers[k]->lb_array;
    float_type *aux_ub_array = fp->layers[k]->ub_array;

    if (fp->layers[k]->activation == RELU) {
      lexpr_replace_relu_bounds<<<num_blocks_relu, num_threads>>>(
          linf_coeff, lsup_coeff, linf_cst, lsup_cst, aux_lb_array,
          aux_ub_array, num_out_neurons_last_layer,
          num_out_neurons_current_layer);
      uexpr_replace_relu_bounds<<<num_blocks_relu, num_threads>>>(
          uinf_coeff, usup_coeff, uinf_cst, usup_cst, aux_lb_array,
          aux_ub_array, num_out_neurons_last_layer,
          num_out_neurons_current_layer);
    }

    hipMalloc((void **)&linf_coeff_tmp, num_out_neurons_last_layer *
                                             num_in_neurons_current_layer *
                                             sizeof(float_type));
    hipMalloc((void **)&lsup_coeff_tmp, num_out_neurons_last_layer *
                                             num_in_neurons_current_layer *
                                             sizeof(float_type));
    hipMalloc((void **)&uinf_coeff_tmp, num_out_neurons_last_layer *
                                             num_in_neurons_current_layer *
                                             sizeof(float_type));
    hipMalloc((void **)&usup_coeff_tmp, num_out_neurons_last_layer *
                                             num_in_neurons_current_layer *
                                             sizeof(float_type));

    coeffs_from_previous_layer<<<num_blocks_linear, num_threads>>>(
        linf_coeff, lsup_coeff, linf_coeff_tmp, lsup_coeff_tmp, aux_inf_coeff,
        aux_sup_coeff, num_out_neurons_last_layer,
        num_out_neurons_current_layer, num_in_neurons_current_layer);
    coeffs_from_previous_layer<<<num_blocks_linear, num_threads>>>(
        uinf_coeff, usup_coeff, uinf_coeff_tmp, usup_coeff_tmp, aux_inf_coeff,
        aux_sup_coeff, num_out_neurons_last_layer,
        num_out_neurons_current_layer, num_in_neurons_current_layer);

    csts_from_previous_layer<<<num_out_neurons_last_layer, 1>>>(
        linf_coeff, lsup_coeff, linf_cst, lsup_cst, linf_cst_tmp, lsup_cst_tmp,
        aux_inf_cst, aux_sup_cst, num_out_neurons_last_layer,
        num_out_neurons_current_layer);
    csts_from_previous_layer<<<num_out_neurons_last_layer, 1>>>(
        uinf_coeff, usup_coeff, uinf_cst, usup_cst, uinf_cst_tmp, usup_cst_tmp,
        aux_inf_cst, aux_sup_cst, num_out_neurons_last_layer,
        num_out_neurons_current_layer);

    std::swap(linf_coeff, linf_coeff_tmp);
    std::swap(lsup_coeff, lsup_coeff_tmp);
    std::swap(linf_cst, linf_cst_tmp);
    std::swap(lsup_cst, lsup_cst_tmp);

    std::swap(uinf_coeff, uinf_coeff_tmp);
    std::swap(usup_coeff, usup_coeff_tmp);
    std::swap(uinf_cst, uinf_cst_tmp);
    std::swap(usup_cst, usup_cst_tmp);

    hipFree(linf_coeff_tmp);
    hipFree(lsup_coeff_tmp);
    hipFree(uinf_coeff_tmp);
    hipFree(usup_coeff_tmp);
  }

  compute_lb_from_expr<<<num_out_neurons_last_layer, 1>>>(
      lb_array, linf_coeff, lsup_coeff, linf_cst, fp->input_inf, fp->input_sup,
      num_out_neurons_last_layer, num_in_neurons_first_layer);
  compute_ub_from_expr<<<num_out_neurons_last_layer, 1>>>(
      ub_array, uinf_coeff, usup_coeff, usup_cst, fp->input_inf, fp->input_sup,
      num_out_neurons_last_layer, num_in_neurons_first_layer);

  hipFree(linf_coeff);
  hipFree(lsup_coeff);
  hipFree(linf_cst);
  hipFree(lsup_cst);

  hipFree(uinf_coeff);
  hipFree(usup_coeff);
  hipFree(uinf_cst);
  hipFree(usup_cst);

  hipFree(linf_coeff_tmp);
  hipFree(lsup_coeff_tmp);
  hipFree(linf_cst_tmp);
  hipFree(lsup_cst_tmp);

  hipFree(uinf_coeff_tmp);
  hipFree(usup_coeff_tmp);
  hipFree(uinf_cst_tmp);
  hipFree(usup_cst_tmp);

  hipDeviceSynchronize();

  auto end = std::chrono::system_clock::now();

  std::chrono::duration<double> elapsed_seconds = end - start;
  std::cout << "elapsed time: " << elapsed_seconds.count() << "s" << std::endl
            << std::endl;
}

void ffn_handle_intermediate_layer(elina_manager_t *man,
                                   elina_abstract0_t *element,
                                   const double **weights, const double *bias,
                                   const size_t num_out_neurons,
                                   const size_t num_in_neurons,
                                   const activation_type_t activation) {
  fppoly_t *fp = fppoly_of_abstract0(element);
  fppoly_add_new_layer(fp, num_out_neurons, num_in_neurons, FFN, activation);

  float_type *inf_coeff = fp->layers[fp->numlayers - 1]->inf_coeff;
  float_type *sup_coeff = fp->layers[fp->numlayers - 1]->sup_coeff;

  float_type *inf_cst = fp->layers[fp->numlayers - 1]->inf_cst;
  float_type *sup_cst = fp->layers[fp->numlayers - 1]->sup_cst;

  layer_create_dense_exprs(inf_coeff, sup_coeff, inf_cst, sup_cst, weights,
                           bias, num_out_neurons, num_in_neurons);

  update_state_using_previous_layers(man, fp, fp->numlayers - 1);
}

void ffn_handle_intermediate_relu_layer(elina_manager_t *man,
                                        elina_abstract0_t *element,
                                        const double **weights,
                                        const double *bias,
                                        const size_t num_out_neurons,
                                        const size_t num_in_neurons) {
  ffn_handle_intermediate_layer(man, element, weights, bias, num_out_neurons,
                                num_in_neurons, RELU);
}

void ffn_handle_intermediate_sigmoid_layer(elina_manager_t *man,
                                           elina_abstract0_t *element,
                                           const double **weights,
                                           const double *bias,
                                           const size_t num_out_neurons,
                                           const size_t num_in_neurons) {
  // ffn_handle_intermediate_layer(man, element, weights, bias, num_out_neurons,
  // num_in_neurons, SIGMOID);
}

void ffn_handle_intermediate_tanh_layer(elina_manager_t *man,
                                        elina_abstract0_t *element,
                                        const double **weights,
                                        const double *bias,
                                        const size_t num_out_neurons,
                                        const size_t num_in_neurons) {
  // ffn_handle_intermediate_layer(man, element, weights, bias, num_out_neurons,
  // num_in_neurons, TANH);
}

__global__ void print_bounds(const float_type *__restrict__ bounds_array,
                             const size_t num_out_neurons) {
  for (size_t i = 0; i < num_out_neurons; i++) {
    printf("out inf number %i is: %g\n", i, bounds_array[i]);
  }
}

void ffn_handle_last_layer(elina_manager_t *man, elina_abstract0_t *element,
                           const double **weights, const double *bias,
                           const size_t num_out_neurons,
                           const size_t num_in_neurons,
                           const bool has_activation,
                           const activation_type_t activation) {
  fppoly_t *fp = fppoly_of_abstract0(element);
  fppoly_internal_t *pr =
      fppoly_init_from_manager(man, ELINA_FUNID_ASSIGN_LINEXPR_ARRAY);

  if (has_activation) {
    fppoly_add_new_layer(fp, num_out_neurons, num_in_neurons, FFN, activation);
  } else {
    fppoly_add_new_layer(fp, num_out_neurons, num_in_neurons, FFN, NONE);
  }

  float_type *inf_coeff = fp->layers[fp->numlayers - 1]->inf_coeff;
  float_type *sup_coeff = fp->layers[fp->numlayers - 1]->sup_coeff;

  float_type *inf_cst = fp->layers[fp->numlayers - 1]->inf_cst;
  float_type *sup_cst = fp->layers[fp->numlayers - 1]->sup_cst;

  layer_create_dense_exprs(inf_coeff, sup_coeff, inf_cst, sup_cst, weights,
                           bias, num_out_neurons, num_in_neurons);

  update_state_using_previous_layers(man, fp, fp->numlayers - 1);

  float_type *lb_array = fp->layers[fp->numlayers - 1]->lb_array;
  float_type *ub_array = fp->layers[fp->numlayers - 1]->ub_array;

  print_bounds<<<1, 1>>>(lb_array, num_out_neurons);
  print_bounds<<<1, 1>>>(ub_array, num_out_neurons);
}

void ffn_handle_last_relu_layer(elina_manager_t *man,
                                elina_abstract0_t *element,
                                const double **weights, const double *bias,
                                const size_t num_out_neurons,
                                const size_t num_in_neurons,
                                const bool has_relu) {
  ffn_handle_last_layer(man, element, weights, bias, num_out_neurons,
                        num_in_neurons, has_relu, RELU);
}

void ffn_handle_last_sigmoid_layer(elina_manager_t *man,
                                   elina_abstract0_t *element,
                                   const double **weights, const double *bias,
                                   const size_t num_out_neurons,
                                   const size_t num_in_neurons,
                                   const bool has_sigmoid) {
  // ffn_handle_last_layer(man, element, weights, bias, num_out_neurons,
  // num_in_neurons, has_sigmoid, SIGMOID);
}

void ffn_handle_last_tanh_layer(elina_manager_t *man,
                                elina_abstract0_t *element,
                                const double **weights, const double *bias,
                                const size_t num_out_neurons,
                                const size_t num_in_neurons,
                                const bool has_tanh) {
  // ffn_handle_last_layer(man, element, weights, bias, num_out_neurons,
  // num_in_neurons, has_tanh, TANH);
}

__global__ void create_sub_expr(float_type *__restrict__ inf_coeff,
                                float_type *__restrict__ sup_coeff,
                                float_type *__restrict__ inf_cst,
                                float_type *__restrict__ sup_cst,
                                const size_t index, const elina_dim_t y,
                                const elina_dim_t x) {
  inf_cst[index] = 0;
  sup_cst[index] = 0;

  for (size_t i = 0; i < 10; i++) {
    inf_coeff[index * 10 + i] = 0.;
    sup_coeff[index * 10 + i] = 0.;
  }

  inf_coeff[index * 10 + y] = -1.;
  sup_coeff[index * 10 + y] = 1.;

  inf_coeff[index * 10 + x] = 1.;
  sup_coeff[index * 10 + x] = -1.;
}

void get_lb_using_previous_layers(elina_manager_t *man,
                                  const fppoly_t *const fp) {
  const size_t numlayers = fp->numlayers;
  fppoly_internal_t *pr =
      fppoly_init_from_manager(man, ELINA_FUNID_ASSIGN_LINEXPR_ARRAY);

  const size_t num_out_neurons_last_layer = 90;

  const size_t num_in_neurons_first_layer = fp->layers[0]->num_in_neurons;

  float_type *lb_dev;
  hipMalloc((void **)&lb_dev, num_out_neurons_last_layer * sizeof(float_type));

  float_type *linf_coeff;
  float_type *lsup_coeff;
  float_type *linf_cst;
  float_type *lsup_cst;

  hipMalloc((void **)&linf_coeff,
             num_out_neurons_last_layer * 10 * sizeof(float_type *));
  hipMalloc((void **)&lsup_coeff,
             num_out_neurons_last_layer * 10 * sizeof(float_type *));
  hipMalloc((void **)&linf_cst,
             num_out_neurons_last_layer * 10 * sizeof(float_type));
  hipMalloc((void **)&lsup_cst,
             num_out_neurons_last_layer * 10 * sizeof(float_type));

  size_t index = 0;

  for (elina_dim_t y = 0; y < 10; y++) {
    for (elina_dim_t x = 0; x < 10; x++) {
      if (y != x) {
        create_sub_expr<<<1, 1>>>(linf_coeff, lsup_coeff, linf_cst, lsup_cst,
                                  index, y, x);
        index++;
      }
    }
  }

  float_type *linf_coeff_tmp;
  float_type *lsup_coeff_tmp;
  float_type *linf_cst_tmp;
  float_type *lsup_cst_tmp;

  hipMalloc((void **)&linf_coeff_tmp,
             num_out_neurons_last_layer * sizeof(float_type *));
  hipMalloc((void **)&lsup_coeff_tmp,
             num_out_neurons_last_layer * sizeof(float_type *));
  hipMalloc((void **)&linf_cst_tmp,
             num_out_neurons_last_layer * sizeof(float_type));
  hipMalloc((void **)&lsup_cst_tmp,
             num_out_neurons_last_layer * sizeof(float_type));

  for (int k = numlayers - 1; k >= 0; k--) {
    const size_t num_out_neurons_current_layer = fp->layers[k]->num_out_neurons;
    const size_t num_in_neurons_current_layer = fp->layers[k]->num_in_neurons;

    const dim3 num_blocks_relu(num_out_neurons_last_layer,
                               num_out_neurons_current_layer / num_threads + 1,
                               1);
    const dim3 num_blocks_linear(num_out_neurons_last_layer,
                                 num_in_neurons_current_layer / num_threads + 1,
                                 1);

    float_type *aux_inf_coeff = fp->layers[k]->inf_coeff;
    float_type *aux_sup_coeff = fp->layers[k]->sup_coeff;

    float_type *aux_inf_cst = fp->layers[k]->inf_cst;
    float_type *aux_sup_cst = fp->layers[k]->sup_cst;

    float_type *aux_lb_array = fp->layers[k]->lb_array;
    float_type *aux_ub_array = fp->layers[k]->ub_array;

    if (fp->layers[k]->activation == RELU) {
      lexpr_replace_relu_bounds<<<num_blocks_relu, num_threads>>>(
          linf_coeff, lsup_coeff, linf_cst, lsup_cst, aux_lb_array,
          aux_ub_array, num_out_neurons_last_layer,
          num_out_neurons_current_layer);
    }

    hipMalloc((void **)&linf_coeff_tmp, num_out_neurons_last_layer *
                                             num_in_neurons_current_layer *
                                             sizeof(float_type));
    hipMalloc((void **)&lsup_coeff_tmp, num_out_neurons_last_layer *
                                             num_in_neurons_current_layer *
                                             sizeof(float_type));

    coeffs_from_previous_layer<<<num_blocks_linear, num_threads>>>(
        linf_coeff, lsup_coeff, linf_coeff_tmp, lsup_coeff_tmp, aux_inf_coeff,
        aux_sup_coeff, num_out_neurons_last_layer,
        num_out_neurons_current_layer, num_in_neurons_current_layer);

    csts_from_previous_layer<<<num_out_neurons_last_layer, 1>>>(
        linf_coeff, lsup_coeff, linf_cst, lsup_cst, linf_cst_tmp, lsup_cst_tmp,
        aux_inf_cst, aux_sup_cst, num_out_neurons_last_layer,
        num_out_neurons_current_layer);

    std::swap(linf_coeff, linf_coeff_tmp);
    std::swap(lsup_coeff, lsup_coeff_tmp);
    std::swap(linf_cst, linf_cst_tmp);
    std::swap(lsup_cst, lsup_cst_tmp);

    hipFree(linf_coeff_tmp);
    hipFree(lsup_coeff_tmp);
  }

  compute_lb_from_expr<<<num_out_neurons_last_layer, 1>>>(
      lb_dev, linf_coeff, lsup_coeff, linf_cst, fp->input_inf, fp->input_sup,
      num_out_neurons_last_layer, num_in_neurons_first_layer);

  hipFree(linf_coeff);
  hipFree(lsup_coeff);
  hipFree(linf_cst);
  hipFree(lsup_cst);

  hipFree(linf_coeff_tmp);
  hipFree(lsup_coeff_tmp);
  hipFree(linf_cst_tmp);
  hipFree(lsup_cst_tmp);

  float_type lb[num_out_neurons_last_layer];
  hipMemcpy(&lb, lb_dev, num_out_neurons_last_layer * sizeof(float_type),
             hipMemcpyDeviceToHost);

  hipFree(lb_dev);

  for (size_t i = 0; i < num_out_neurons_last_layer; i++) {
    if (lb[i] < 0) {
      results[i] = true;
    } else {
      results[i] = false;
    }
  }
}

bool is_greater(elina_manager_t *man, elina_abstract0_t *element,
                const elina_dim_t y, const elina_dim_t x) {
  const fppoly_t *fp = fppoly_of_abstract0(element);
  fppoly_internal_t *pr =
      fppoly_init_from_manager(man, ELINA_FUNID_ASSIGN_LINEXPR_ARRAY);

  if (!results_calculated) {
    get_lb_using_previous_layers(man, fp);
    results_calculated = true;

    return results[0];
  } else {
    bool result = results[output_counter];
    output_counter++;

    return result;
  }
}

void device_layer_create_sparse_exprs(
    float_type *inf_coeff, float_type *sup_coeff, float_type *inf_cst,
    float_type *sup_cst, const double *filter_weights,
    const double *filter_bias, const size_t *input_size,
    const size_t *output_size, const size_t *filter_size, const size_t *strides,
    const long int pad_top, const long int pad_left, const size_t num_pixels) {
  const size_t num_out_neurons =
      output_size[0] * output_size[1] * output_size[2];

  double *dense_coeff =
      (double *)calloc(num_out_neurons * num_pixels, sizeof(double));
  double *bias = (double *)calloc(num_out_neurons, sizeof(double));

  for (size_t out_x = 0; out_x < output_size[0]; out_x++) {
    for (size_t out_y = 0; out_y < output_size[1]; out_y++) {
      for (size_t out_z = 0; out_z < output_size[2]; out_z++) {
        const size_t mat_x = out_x * output_size[1] * output_size[2] +
                             out_y * output_size[2] + out_z;

        for (size_t x_shift = 0; x_shift < filter_size[0]; x_shift++) {
          for (size_t y_shift = 0; y_shift < filter_size[1]; y_shift++) {
            for (size_t inp_z = 0; inp_z < input_size[2]; inp_z++) {
              const long int x_val = out_x * strides[0] + x_shift - pad_top;
              const long int y_val = out_y * strides[1] + y_shift - pad_left;

              if ((y_val < 0) || (y_val >= (long int)input_size[1])) {
                continue;
              }

              if ((x_val < 0) || (x_val >= (long int)input_size[0])) {
                continue;
              }

              const size_t mat_y = x_val * input_size[1] * input_size[2] +
                                   y_val * input_size[2] + inp_z;

              if (mat_y >= num_pixels) {
                continue;
              }

              const size_t filter_index =
                  x_shift * filter_size[1] * input_size[2] * output_size[2] +
                  y_shift * input_size[2] * output_size[2] +
                  inp_z * output_size[2] + out_z;
              dense_coeff[mat_x * num_pixels + mat_y] =
                  filter_weights[filter_index];
            }
          }
        }

        bias[mat_x] = filter_bias[out_z];
      }
    }
  }

  double *dense_coeff_dev;
  double *bias_dev;

  hipMalloc((void **)&dense_coeff_dev,
             num_out_neurons * num_pixels * sizeof(double));
  hipMalloc((void **)&bias_dev, num_out_neurons * sizeof(double));

  hipMemcpy(dense_coeff_dev, dense_coeff,
             num_out_neurons * num_pixels * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(bias_dev, bias, num_out_neurons * sizeof(double),
             hipMemcpyHostToDevice);

  device_layer_create_dense_expr<<<num_out_neurons, 1>>>(
      inf_coeff, sup_coeff, inf_cst, sup_cst, dense_coeff_dev, bias_dev,
      num_out_neurons, num_pixels);

  hipFree(dense_coeff_dev);
  hipFree(bias_dev);

  free(dense_coeff);
  free(bias);
}

void layer_create_sparse_exprs(fppoly_t *const fp, const double *filter_weights,
                               const double *filter_bias,
                               const size_t *input_size,
                               const size_t *filter_size,
                               const size_t num_filters, const size_t *strides,
                               const bool is_valid_padding,
                               const bool has_bias) {
  const size_t num_pixels = input_size[0] * input_size[1] * input_size[2];

  size_t output_size[3];

  if (is_valid_padding) {
    output_size[0] =
        ceil((double)(input_size[0] - filter_size[0] + 1) / (double)strides[0]);
    output_size[1] =
        ceil((double)(input_size[1] - filter_size[1] + 1) / (double)strides[1]);
  } else {
    output_size[0] = ceil((double)input_size[0] / (double)strides[0]);
    output_size[1] = ceil((double)input_size[1] / (double)strides[1]);
  }

  output_size[2] = num_filters;

  const size_t num_out_neurons =
      output_size[0] * output_size[1] * output_size[2];
  fppoly_add_new_layer(fp, num_out_neurons, num_pixels, CONV, RELU);

  float_type *inf_coeff = fp->layers[fp->numlayers - 1]->inf_coeff;
  float_type *sup_coeff = fp->layers[fp->numlayers - 1]->sup_coeff;

  float_type *inf_cst = fp->layers[fp->numlayers - 1]->inf_cst;
  float_type *sup_cst = fp->layers[fp->numlayers - 1]->sup_cst;

  long int pad_along_height = 0;
  long int pad_along_width = 0;
  long int pad_top = 0;
  long int pad_left = 0;

  if (!is_valid_padding) {
    if (input_size[0] % strides[0] == 0) {
      const long int tmp = filter_size[0] - strides[0];
      pad_along_height = max(tmp, long(0));
    } else {
      const long int tmp = filter_size[0] - (input_size[0] % strides[0]);
      pad_along_height = max(tmp, long(0));
    }

    if (input_size[1] % strides[1] == 0) {
      const long int tmp = filter_size[1] - strides[1];
      pad_along_width = max(tmp, long(0));
    } else {
      const long int tmp = filter_size[1] - (input_size[1] % strides[1]);
      pad_along_width = max(tmp, long(0));
    }

    pad_top = pad_along_height / 2;
    pad_left = pad_along_width / 2;
  }

  const size_t size =
      filter_size[0] * filter_size[1] * input_size[2] * output_size[2];

  double *filter_weights_tmp = (double *)malloc(size * sizeof(double));
  double *filter_bias_tmp = (double *)calloc(output_size[2], sizeof(double));

  size_t *input_size_tmp = (size_t *)malloc(3 * sizeof(size_t));
  size_t *output_size_tmp = (size_t *)malloc(3 * sizeof(size_t));
  size_t *filter_size_tmp = (size_t *)malloc(2 * sizeof(size_t));
  size_t *strides_tmp = (size_t *)malloc(2 * sizeof(size_t));

  hipMemcpy(filter_weights_tmp, filter_weights, size * sizeof(double),
             hipMemcpyHostToHost);

  if (has_bias) {
    hipMemcpy(filter_bias_tmp, filter_bias, output_size[2] * sizeof(double),
               hipMemcpyHostToHost);
  }

  hipMemcpy(input_size_tmp, input_size, 3 * sizeof(size_t),
             hipMemcpyHostToHost);
  hipMemcpy(output_size_tmp, output_size, 3 * sizeof(size_t),
             hipMemcpyHostToHost);
  hipMemcpy(filter_size_tmp, filter_size, 2 * sizeof(size_t),
             hipMemcpyHostToHost);
  hipMemcpy(strides_tmp, strides, 2 * sizeof(size_t), hipMemcpyHostToHost);

  device_layer_create_sparse_exprs(
      inf_coeff, sup_coeff, inf_cst, sup_cst, filter_weights_tmp,
      filter_bias_tmp, input_size_tmp, output_size_tmp, filter_size_tmp,
      strides_tmp, pad_top, pad_left, num_pixels);

  free(filter_weights_tmp);
  free(filter_bias_tmp);

  free(input_size_tmp);
  free(output_size_tmp);
  free(filter_size_tmp);
  free(strides_tmp);
}

void conv_handle_first_layer(elina_manager_t *man, elina_abstract0_t *element,
                             const double *filter_weights,
                             const double *filter_bias,
                             const size_t *input_size,
                             const size_t *filter_size,
                             const size_t num_filters, const size_t *strides,
                             const bool is_valid_padding, const bool has_bias) {
  fppoly_t *const fp = fppoly_of_abstract0(element);
  fp->layers = (layer_t **)malloc(20 * sizeof(layer_t *));

  layer_create_sparse_exprs(fp, filter_weights, filter_bias, input_size,
                            filter_size, num_filters, strides, is_valid_padding,
                            has_bias);

  float_type *inf_coeff = fp->layers[0]->inf_coeff;
  float_type *sup_coeff = fp->layers[0]->sup_coeff;

  float_type *inf_cst = fp->layers[0]->inf_cst;
  float_type *sup_cst = fp->layers[0]->sup_cst;

  layer_compute_bounds_from_exprs(
      inf_coeff, sup_coeff, inf_cst, sup_cst, fp->layers[0]->lb_array,
      fp->layers[0]->ub_array, fp->input_inf, fp->input_sup,
      fp->layers[0]->num_out_neurons, fp->layers[0]->num_in_neurons);
}

void conv_handle_intermediate_relu_layer(
    elina_manager_t *man, elina_abstract0_t *element,
    const double *filter_weights, const double *filter_bias,
    const size_t *input_size, const size_t *filter_size,
    const size_t num_filters, const size_t *strides,
    const bool is_valid_padding, const bool has_bias) {
  fppoly_t *const fp = fppoly_of_abstract0(element);

  layer_create_sparse_exprs(fp, filter_weights, filter_bias, input_size,
                            filter_size, num_filters, strides, is_valid_padding,
                            has_bias);

  update_state_using_previous_layers(man, fp, fp->numlayers - 1);
}

void free_layer(layer_t *layer) {
  hipFree(layer->inf_coeff);
  hipFree(layer->sup_coeff);
  hipFree(layer->inf_cst);
  hipFree(layer->sup_cst);

  layer->inf_coeff = nullptr;
  layer->sup_coeff = nullptr;
  layer->inf_cst = nullptr;
  layer->sup_cst = nullptr;

  hipFree(layer->lb_array);
  hipFree(layer->ub_array);

  layer->lb_array = nullptr;
  layer->ub_array = nullptr;

  free(layer);
  layer = nullptr;
}

void fppoly_free(elina_manager_t *man, fppoly_t *fp) {
  for (size_t i = 0; i < fp->numlayers; i++) {
    free_layer(fp->layers[i]);
  }

  free(fp->layers);
  fp->layers = nullptr;

  hipFree(fp->input_inf);
  fp->input_inf = nullptr;
  hipFree(fp->input_sup);
  fp->input_sup = nullptr;

  free(fp);
  fp = nullptr;
}

void layer_print(const layer_t *layer) {
  // neurons_print<<<1, 1>>>(layer->neurons, layer->num_out_neurons);
}

void fppoly_fprint(FILE *const stream, elina_manager_t *man,
                   const fppoly_t *const fp, const char **name_of_dim) {
  for (size_t i = 0; i < fp->numlayers; i++) {
    printf("layer: %zu\n", i);
    layer_print(fp->layers[i]);
  }
}
